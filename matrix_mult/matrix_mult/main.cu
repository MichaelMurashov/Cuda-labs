#include <iostream>
#include <time.h>
#include <omp.h>


#include "hip/hip_runtime.h"

using namespace std;

const int block_size = 16;

__global__
void shared_kernel(int n, const float *a, const float *b, float *c) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = n * block_size * by;
    int aEnd = aBegin + n - 1;
    int bBegin = block_size * bx;
    int bStep = block_size * n;

    int aStep = block_size;

    float sum = 0.0f;

    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
        __shared__ float as[block_size][block_size];
        __shared__ float bs[block_size][block_size];

        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];

        __syncthreads();

        for (int k = 0; k < block_size; k++)
            sum += as[ty][k] * bs[k][tx];

        __syncthreads();
    }

    int ic = n * block_size * by + block_size * bx;

    c[ic + n * ty + tx] = sum;
}

float sharedMultMat(int n, const float *dev_a, const float *dev_b, float *dev_c) {
    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    clock_t start = clock();

    shared_kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

__global__
void kernel(int n, const float *a, const float *b, float *c) {
    int   bx = blockIdx.x;
    int   by = blockIdx.y;
    int   tx = threadIdx.x;
    int   ty = threadIdx.y;

    float sum = 0.0f;

    int   ia = n * block_size * by + n * ty;
    int   ib = block_size * bx + tx;

    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k*n];

    int ic = n * block_size * by + block_size * bx;
    c[ic + n * ty + tx] = sum;
}

float multMat(int n, const float *dev_a, const float *dev_b, float *dev_c) {
    //int num_block = (n + block_size - 1) / block_size;

    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    clock_t start = clock();

    kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

float cpuMultMat(int n, const float *a, const float *b, float*c) {
    int i, j, k;
    int NestedThreadsNum = 2;

    omp_set_nested(true);
    omp_set_num_threads(NestedThreadsNum);

    clock_t start = clock();

    #pragma omp parallel for private (j, k)
    for (i = 0; i < n; i++)
        #pragma omp parallel for private (k)
        for (j = 0; j < n; j++)
            for (k = 0; k < n; k++)
                c[i * n + j] += a[i * n + k] * b[k * n + j];

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

int main() {
    const int n = 64;

    float *a = new float[n * n], *b = new float[n * n], *c = new float[n * n];
    float *resultGPU = new float[n * n], *resultSharedGPU = new float[n * n];

    float *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    for (int i = 0; i < n * n; i++) {
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * n * sizeof(float), hipMemcpyHostToDevice);

    float cpuTime = cpuMultMat(n, a, b, c);

    float gpuTime = multMat(n, dev_a, dev_b, dev_c);

    hipMemcpy(resultGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    float sharedGpuTime = sharedMultMat(n, dev_a, dev_b, dev_c);

    hipMemcpy(resultSharedGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    //for (int i = 0; i < n * n; i++)
    //    cout << c[i] << ' ';
    //cout << endl;

    //for (int i = 0; i < n * n; i++)
    //    cout << resultGPU[i] << ' ';
    //cout << endl;

    for (int i = 0; i < n * n; i++)
        if (c[i] != resultGPU[i]) {
            cout << "Matrixs cpu and gpu are not equal!" << endl;
            break;
        }

    for (int i = 0; i < n * n; i++)
        if (c[i] != resultSharedGPU[i]) {
            cout << "Matrixs cpu and shared_gpu are not equal!" << endl;
            break;
        }

    cout << "gpu time: " << gpuTime << endl
        << "shared gpu time: " << sharedGpuTime << endl
        << "cpu time: " << cpuTime << endl;

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    free(a); free(b); free(c);
    system("pause");

    return 0;
}
