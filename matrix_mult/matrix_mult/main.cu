#include <iostream>
#include <time.h>
#include <omp.h>
#include <iomanip>


#include "hip/hip_runtime.h"


using namespace std;

#define block_size 64

__global__
void shared_kernel(int n, const float *a, const float *b, float *c) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = n * block_size * by;
    int aEnd = aBegin + n - 1;
    int bBegin = block_size * bx;
    int bStep = block_size * n;

    int aStep = block_size;

    float sum = 0.0f;

    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
        __shared__ float as[block_size][block_size];
        __shared__ float bs[block_size][block_size];

        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];

        __syncthreads();

        for (int k = 0; k < block_size; k++)
            sum += as[ty][k] * bs[k][tx];

        __syncthreads();
    }

    int ic = n * block_size * by + block_size * bx;

    c[ic + n * ty + tx] = sum;
}

float sharedMultMat(int n, const float *dev_a, const float *dev_b, float *dev_c) {
    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    clock_t start = clock();

    shared_kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

__global__
void kernel(int n, const float *a, const float *b, float *c) {
    int   bx = blockIdx.x;
    int   by = blockIdx.y;
    int   tx = threadIdx.x;
    int   ty = threadIdx.y;

    float sum = 0.0f;

    int   ia = n * block_size * by + n * ty;
    int   ib = block_size * bx + tx;

    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k*n];

    int ic = n * block_size * by + block_size * bx;
    c[ic + n * ty + tx] = sum;
}

float multMat(int n, const float *dev_a, const float *dev_b, float *dev_c) {
    //int num_block = (n + block_size - 1) / block_size;

    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    clock_t start = clock();

    kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

float cpuMultMat(int n, const float *a, const float *b, float*c) {
    int i, j, k;

    clock_t start = clock();

    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            float elem = 0.0f;
            for (k = 0; k < n; k++)
                elem += a[i * n + k] * b[k * n + j];
            c[i * n + j] = elem;
        }

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

int main() {
    const int n = 2048;
    const float e = 0.00001;

    float *a = new float[n * n], *b = new float[n * n], *c = new float[n * n];
    float *resultGPU = new float[n * n], *resultSharedGPU = new float[n * n];

    float *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    for (int i = 0; i < n * n; i++) {
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * n * sizeof(float), hipMemcpyHostToDevice);

    float cpuTime = cpuMultMat(n, a, b, c);

    cout << fixed << setprecision(8) 
        << "cpu time: " << cpuTime << endl;

    float gpuTime = multMat(n, dev_a, dev_b, dev_c);

    hipMemcpy(resultGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    float sharedGpuTime = sharedMultMat(n, dev_a, dev_b, dev_c);

    hipMemcpy(resultSharedGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    //for (int i = 0; i < n * n; i++)
    //    cout << c[i] << ' ';
    //cout << endl;

    //for (int i = 0; i < n * n; i++)
    //    cout << resultGPU[i] << ' ';
    //cout << endl;

    /*for (int i = 0; i < n * n; i++)
        if (std::abs(c[i] - resultGPU[i]) < e) {
            cout << "Matrixs cpu and gpu are not equal!" << endl;
            break;
        }

    for (int i = 0; i < n * n; i++)
        if (std::abs(c[i] - resultSharedGPU[i]) < e) {
            cout << "Matrixs cpu and shared_gpu are not equal!" << endl;
            break;
        }*/

    cout << fixed << "gpu time: " << setprecision(8) << gpuTime << endl
        << "shared gpu time: " << sharedGpuTime << endl
        << "cpu time: " << cpuTime << endl;

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    delete[]a; free(b); free(c);
    system("pause");

    return 0;
}
