#include <iostream>
#include <time.h>
#include <omp.h>


#include "hip/hip_runtime.h"

using namespace std;

const int block_size = 16;

__global__
void kernel(int n, const float *a, const float *b, float *c) {
    int   bx = blockIdx.x;     // block index
    int   by = blockIdx.y;
    int   tx = threadIdx.x;        // thread index
    int   ty = threadIdx.y;
    float sum = 0.0f;           // computed subelement
    int   ia = n * block_size * by + n * ty;   // a [i][0]
    int   ib = block_size * bx + tx;

    // Multiply the two matrices together;
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k*n];

    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int ic = n * block_size * by + block_size * bx;

    c[ic + n * ty + tx] = sum;
}

float multMat(int n, const float *dev_a, const float *dev_b, float *dev_c) {
    //int num_block = (n + block_size - 1) / block_size;

    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    clock_t start = clock();

    kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

float cpuMultMat(int n, const float *a, const float *b, float*c) {
    int i, j, k;
    int NestedThreadsNum = 2;

    omp_set_nested(true);
    omp_set_num_threads(NestedThreadsNum);

    clock_t start = clock();

    #pragma omp parallel for private (j, k)
    for (i = 0; i < n; i++)
        #pragma omp parallel for private (k)
        for (j = 0; j < n; j++)
            for (k = 0; k < n; k++)
                c[i * n + j] += a[i * n + k] * b[k * n + j];

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

int main() {
    const int n = 64;

    float *a = new float[n * n], *b = new float[n * n], *c = new float[n * n];
    float *resultGPU = new float[n * n];

    float *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    for (int i = 0; i < n * n; i++) {
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * n * sizeof(float), hipMemcpyHostToDevice);

    float cpuTime = cpuMultMat(n, a, b, c);

    float gpuTime = multMat(n, dev_a, dev_b, dev_c);

    hipMemcpy(resultGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    //for (int i = 0; i < n * n; i++)
    //    cout << c[i] << ' ';
    //cout << endl;

    //for (int i = 0; i < n * n; i++)
    //    cout << resultGPU[i] << ' ';
    //cout << endl;

    for (int i = 0; i < n * n; i++)
        if (c[i] != resultGPU[i]) {
            cout << "Matrixs are not equal!" << endl;
            break;
        }

    cout << "gpu time: " << gpuTime << endl
        << "cpu time: " << cpuTime << endl;

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    free(a); free(b); free(c);
    system("pause");

    return 0;
}
