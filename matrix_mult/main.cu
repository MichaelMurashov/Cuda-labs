#include <iostream>
#include <time.h>
#include <omp.h>
#include <iomanip>
#include <cmath>
#include <stdio.h>


#include "hip/hip_runtime.h"


using namespace std;

#define block_size 16

__global__
void shared_kernel(int n, const float *a, const float *b, float *c) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = n * block_size * by;
    int aEnd = aBegin + n - 1;
    int bBegin = block_size * bx;
    int bStep = block_size * n;

    int aStep = block_size;

    float sum = 0.0f;

    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
        __shared__ float as[block_size][block_size];
        __shared__ float bs[block_size][block_size];

        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];

        __syncthreads();

        for (int k = 0; k < block_size; k++)
            sum += as[ty][k] * bs[k][tx];

        __syncthreads();
    }

    int ic = n * block_size * by + block_size * bx;

    c[ic + n * ty + tx] = sum;
}

float sharedMultMat(int n, const float *a, const float *b, const float *c, float *resultSharedGPU) {
    float *dev_a, *dev_b, *dev_c;

    dim3 threads(block_size, block_size);
    dim3 blocks(n / threads.x, n / threads.y);

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    clock_t start = clock();

    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * n * sizeof(float), hipMemcpyHostToDevice);

    shared_kernel <<< blocks, threads >>> (n, dev_a, dev_b, dev_c);

    hipMemcpy(resultSharedGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    clock_t finish = clock();

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

__global__
void kernel_t(int n, const float *a, const float *b, float *c) {

   // printf("from kernel\n");
    int   bx = blockIdx.x;
    int   by = blockIdx.y;
    int   tx = threadIdx.x;
    int   ty = threadIdx.y;

    float sum = 0.0f;

    int   ia = n * block_size * by + n * ty;
    int   ib = block_size * bx + tx;

    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k*n];

    int ic = n * block_size * by + block_size * bx;
    c[ic + n * ty + tx] = sum;
}

float multMat(int n, const float *a, const float *b, const float *c, float *resultGPU) {
    float *dev_a, *dev_b, *dev_c;

    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid(n / dimBlock.x, n / dimBlock.y);

    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_c, n * n * sizeof(float));

    clock_t _start = clock();

    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * n * sizeof(float), hipMemcpyHostToDevice);

    kernel_t <<< dimGrid, dimBlock >>> (n, dev_a, dev_b, dev_c);

    hipMemcpy(resultGPU, dev_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    clock_t _finish = clock();

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return (float)(_finish - _start) / CLOCKS_PER_SEC;
}

float cpuMultMat(int n, const float *a, const float *b, float*c) {
    int i, j, k;

    clock_t start = clock();

    #pragma omp parallel for private (j, k)
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            float elem = 0.0f;
            for (k = 0; k < n; k++)
                elem += a[i * n + k] * b[k * n + j];
            c[i * n + j] = elem;
        }

    clock_t finish = clock();

    return (float)(finish - start) / CLOCKS_PER_SEC;
}

int main() {
    const int n = 256;
    const float e = 1e-5;

    float *a = new float[n * n], *b = new float[n * n], *c = new float[n * n];
    float *resultGPU = new float[n * n], *resultSharedGPU = new float[n * n];

    for (int i = 0; i < n * n; i++) {
        a[i] = b[i] = cos(i);
        c[i] = 0;
    }

    float gpuTime = multMat(n, a, b, c, resultGPU);

    float sharedGpuTime = sharedMultMat(n, a, b, c, resultSharedGPU);

    float cpuTime = cpuMultMat(n, a, b, c);

//    for (int i = 0; i < n * n; i++)
//        cout << c[i] << "|" << resultGPU[i] << "|" << resultsharedGPU[i] << endl;

    for (int i = 0; i < n * n; i++)
        if (std::abs(c[i] - resultGPU[i]) > e) {
            cout << "Matrixs cpu and gpu are not equal!" << endl;
            break;
        }

    for (int i = 0; i < n * n; i++)
        if (std::abs(c[i] - resultSharedGPU[i]) > e) {
            cout << "Matrixs cpu and shared_gpu are not equal!" << endl;
            break;
        }

    cout << fixed << setw(19) << left << "gpu time: " << setprecision(6) << gpuTime << endl
        << "optimize gpu time: " << sharedGpuTime << endl
        << setw(19) << "cpu time: " << cpuTime << endl;

    delete[]a; delete[]b; delete[]c;

    return 0;
}

